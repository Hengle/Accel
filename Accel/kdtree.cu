#include "hip/hip_runtime.h"
#include "kdtree.h"


kdtree::kdtree(mesh* m){
	this->m_mesh = m;
	
	this->cudaTris = NULL;
	hipMalloc((void**)&cudaTris, m->numTris*sizeof(glm::vec3));
	hipMemcpy( cudaTris, m->tris, m->numTris*sizeof(glm::vec3), hipMemcpyHostToDevice);

	this->cudaVerts = NULL;
	hipMalloc((void**)&cudaVerts, m->numVerts*sizeof(glm::vec3));
	hipMemcpy( cudaVerts, m->verts, m->numVerts*sizeof(glm::vec3), hipMemcpyHostToDevice);

	//debug
	this->boundingBoxes = new boundingBox[m->numTris];
	this->cudaBoundingBoxes = NULL;
	hipMalloc((void**)&cudaBoundingBoxes, m->numTris*sizeof(boundingBox));
	hipMemcpy( cudaBoundingBoxes, this->boundingBoxes, m->numTris*sizeof(boundingBox), hipMemcpyHostToDevice);
}

kdtree::~kdtree(){
	
	hipFree(this->cudaTris);
	hipFree(this->cudaVerts);

	delete [] this->boundingBoxes;
	hipFree(this->cudaBoundingBoxes);
}

void kdtree::construct(){
	perTriBoundingBox();
}

__global__ 
void perTriBoundingBoxKernel(int numTris, glm::vec3* tris, glm::vec3* verts, boundingBox* bbs){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index<numTris){
		glm::vec3 tri = tris[index];
		int ix = int(tri.x);
		int iy = int(tri.y);
		int iz = int(tri.z);

		glm::vec3 p1 = verts[ix];
		glm::vec3 p2 = verts[iy];
		glm::vec3 p3 = verts[iz];

		float maxX = glm::max(p1.x, p2.x);
		maxX = glm::max(maxX, p3.x);

		float maxY = glm::max(p1.y, p2.y);
		maxY = glm::max(maxY, p3.y);

		float maxZ = glm::max(p1.z, p2.z);
		maxZ = glm::max(maxZ, p3.z);

		float minX = glm::min(p1.x, p2.x);
		minX = glm::min(minX, p3.x);

		float minY = glm::min(p1.y, p2.y);
		minY = glm::min(minY, p3.y);

		float minZ = glm::min(p1.z, p2.z);
		minZ = glm::min(minZ, p3.z);

		bbs[index].max = glm::vec3(maxX,maxY,maxZ);
		bbs[index].min = glm::vec3(minX,minY,minZ);
	}
}

void kdtree::perTriBoundingBox(){
	dim3 threadsPerBlock(64);
	dim3 fullBlocksPerGrid(m_mesh->numTris/8+1);

	perTriBoundingBoxKernel<<<fullBlocksPerGrid, threadsPerBlock>>>
		(this->m_mesh->numTris, this->cudaTris, this->cudaVerts, this->cudaBoundingBoxes);

	hipMemcpy( this->boundingBoxes, cudaBoundingBoxes, this->m_mesh->numTris*sizeof(boundingBox), hipMemcpyDeviceToHost);
}