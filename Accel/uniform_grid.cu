#include "hip/hip_runtime.h"
#include "uniform_grid.h"

using namespace std;

glm::vec3 *cudapositions;
pair<int, int>* cudagrid;
int* cudaneighbors;
int* cudaids, *cudacellIds, *cudapIds;
int* cudanumNeighbors;

bool checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
	cout<<"Cuda error at "<<msg<<": "<<hipGetErrorString(err)<<endl;
	cout<<endl;
	return false;
  }
  return true;
} 

__device__ 
float lengthSquared(glm::vec3 p){
    return p.x*p.x + p.y*p.y + p.z*p.z;
}

__device__ 
int hashParticle(glm::vec3 p, glm::vec3 gridSize, float h){
	int x = p.x/h;
	int y = p.y/h;
	int z = p.z/h;
	return x + y*gridSize.x + z*gridSize.x*gridSize.y;
}

__global__ 
void hashParticlesToGridKernel(int numParticles, int* cellIds, int* pIds, glm::vec3* positions, glm::vec3 gridSize, int* ids, float h){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index<numParticles){
		index = ids[index];
		cellIds[index] = hashParticle(positions[index], gridSize, h);
		pIds[index] = index;
	}
}

__global__ 
void resetGrid(int numGridCells, pair<int,int>* grid){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index<numGridCells){
		grid[index].first = -1;
		grid[index].second = -1;
	}
}


__global__ 
void setGridValuesKernel(int numParticles, int numGridCells, pair<int,int>* grid, int* cellIds){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index<numParticles){
		int cellId = cellIds[index];
		if (index==0){
			grid[cellId].first = index;
		}
		else{
			if (cellId == cellIds[index-1]){
				if (index > grid[cellId].second){
					grid[cellId].second = index;
				}
			}
			else{
				grid[cellId].first = index;
			}
		}
	}
}

__global__ 
void findNeighborsUsingGridKernel(int numParticles, float h, int maxNeighbors, glm::vec3* positions, 
								  int* neighbors, int* numNeighbors, pair<int,int>* grid, int* pIds, 
								  int numGridCells, int* ids, glm::vec3 gridSize, int* cellIds){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index<numParticles){
		index = ids[index];
		numNeighbors[index] = 0;

		int cellIdOriginal = hashParticle(positions[index], gridSize, h);
		if (cellIdOriginal<0 || cellIdOriginal>numGridCells-1) return;
		//search for neighbors in 3x3x3 neighbor grid cells
		//favors closer neighboring cells
		for (int cell=0; cell<27; cell++){
			int cellId1 = grid[cellIdOriginal].first;
			int cellId2 = grid[cellIdOriginal].second;
			
			if (numNeighbors[index]>=maxNeighbors){
				break;
			}

			int cellId = cellIdOriginal;

			if (cell==1){ //+x
				cellId = cellId+1;
			}
			else if (cell==2){ //-x
				cellId = cellId-1;
			}
			else if (cell==3){ //+y
				cellId = cellId+gridSize.x;
			}
			else if (cell==4){ //+z
				cellId = cellId+gridSize.x*gridSize.y;
			}
			else if (cell==5){ //-z
				cellId = cellId-gridSize.x*gridSize.y;
			}
			else if (cell==6){ //-y
				cellId = cellId-gridSize.x;
			}
			else if (cell==7){ //+x +y
				cellId = cellId+gridSize.x+1;
			}
			else if (cell==8){ //+x +z
				cellId = cellId+gridSize.x*gridSize.y+1;
			}
			else if (cell==9){ //+x -y
				cellId = cellId-gridSize.x+1;
			}
			else if (cell==10){ //+x -z
				cellId = cellId-gridSize.x*gridSize.y+1;
			}
			else if (cell==11){ //-x +y
				cellId = cellId+gridSize.x-1;
			}
			else if (cell==12){ //-x -y
				cellId = cellId-gridSize.x-1;
			}
			else if (cell==13){ //-x +z
				cellId = cellId+gridSize.x*gridSize.y-1;
			} 
			else if (cell==14){ //-x -z
				cellId = cellId-gridSize.x*gridSize.y-1;
			}
			else if (cell==15){ //+y +z
				cellId = cellId+gridSize.x+gridSize.x*gridSize.y;
			}
			else if (cell==16){ //+y -z
				cellId = cellId+gridSize.x-gridSize.x*gridSize.y;
			}
			else if (cell==17){ //-y +z
				cellId = cellId-gridSize.x+gridSize.x*gridSize.y;
			}
			else if (cell==18){ //-y -z
				cellId = cellId-gridSize.x-gridSize.x*gridSize.y;
			}
			else if (cell==19){ //+x +y +z
				cellId = cellId+gridSize.x*gridSize.y+gridSize.x+1;
			}
			else if (cell==20){ //+x -y +z
				cellId = cellId+gridSize.x*gridSize.y-gridSize.x+1;
			}
			else if (cell==21){ //+x +y -z
				cellId = cellId-gridSize.x*gridSize.y+gridSize.x+1;
			}
			else if (cell==22){ //+x -y -z
				cellId = cellId-gridSize.x*gridSize.y-gridSize.x+1;
			}
			else if (cell==23){ //-x +y +z
				cellId = cellId+gridSize.x*gridSize.y+gridSize.x-1;
			}
			else if (cell==24){ //-x -y +z
				cellId = cellId+gridSize.x*gridSize.y-gridSize.x-1;
			}
			else if (cell==25){ //-x +y -z
				cellId = cellId-gridSize.x*gridSize.y+gridSize.x-1;
			}
			else if (cell==26){ //-x -y -z
				cellId = cellId-gridSize.x*gridSize.y-gridSize.x-1;
			}

			//sanity check
			if (cellId<0 || cellId>numGridCells-1) continue;

			cellId1 = grid[cellId].first;
			if (cellId1==-1) continue;
			cellId2 = grid[cellId].second;
			if (cellId2==-1) cellId2=cellId1;
			
			if (cellId1>numParticles-1 || cellId1<0 || cellId2>numParticles-1 || cellId2<0) continue;

			int jid = ids[pIds[cellId1]];
			if (lengthSquared(positions[index]-positions[jid])<h*h && numNeighbors[index]<maxNeighbors){
				int nid = index*maxNeighbors+numNeighbors[index];
				neighbors[nid]=jid;
				numNeighbors[index]+=1;
			}

			for (int i=cellId1; i<cellId2+1; i+=1){
				int jid = ids[pIds[i]];
				if (lengthSquared(positions[index]-positions[jid])<h*h && numNeighbors[index]<maxNeighbors){
					int nid = index*maxNeighbors+numNeighbors[index];
					neighbors[nid]=jid;
					numNeighbors[index]+=1;
				}
			}
		}
	}
}

__global__
void findNeighborsKernel(int numParticles, glm::vec3* positions, int* neighbors, int* numNeighbors, float h, int maxNeighbors, int* ids){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index<numParticles){
		index = ids[index];
		numNeighbors[index] = 0;
		for (int j=0; j<numParticles; j++){
			if (lengthSquared(positions[index]-positions[j])<h*h && numNeighbors[index]<maxNeighbors){
				neighbors[index*maxNeighbors+numNeighbors[index]]=ids[j];
				numNeighbors[index]+=1;
			}
		}
	}
}

__global__
void resetNumNeighbors(int numParticles, int* numNeighbors){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index<numParticles){
		numNeighbors[index] = 0;
	}
}

__global__
void hello(int *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

void initCuda(int numParticles, int* ids, glm::vec3* positions, int maxNeighbors, glm::vec3 gridSize){
	cudapositions = NULL;
	hipMalloc((void**)&cudapositions, numParticles*sizeof(glm::vec3));
	hipMemcpy( cudapositions, positions, numParticles*sizeof(glm::vec3), hipMemcpyHostToDevice);

	cudaneighbors = NULL;
	hipMalloc((void**)&cudaneighbors, numParticles*maxNeighbors*sizeof(int*));

	cudanumNeighbors = NULL;
	hipMalloc((void**)&cudanumNeighbors, numParticles*sizeof(int));
	hipMemcpy( cudanumNeighbors, ids, numParticles*sizeof(int), hipMemcpyHostToDevice);
	
	cudaids = NULL;
	hipMalloc((void**)&cudaids, numParticles*sizeof(int));
	hipMemcpy( cudaids, ids, numParticles*sizeof(int), hipMemcpyHostToDevice);

	cudacellIds = NULL;
	hipMalloc((void**)&cudacellIds, numParticles*sizeof(int));

	cudapIds = NULL;
	hipMalloc((void**)&cudapIds, numParticles*sizeof(int));

	cudagrid = NULL;
	hipMalloc((void**)&cudagrid, int(gridSize.x*gridSize.y*gridSize.z)*sizeof(pair<int,int>));
}

void findNeighbors(int numParticles, int maxNeighbors, glm::vec3 gridSize, float h){
	dim3 threadsPerBlock(64);
	dim3 fullBlocksPerGrid(numParticles/8+1);

	dim3 threadsPerBlockGrid(64);
	dim3 fullBlocksPerGridGrid(int(gridSize.x*gridSize.y*gridSize.z)/8+1);

	hashParticlesToGridKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(numParticles, cudacellIds, cudapIds, cudapositions, gridSize, cudaids, h);
	hipDeviceSynchronize();
	checkCUDAError("hasing particles");

	thrust::device_ptr<int> thrustCellIds = thrust::device_pointer_cast(cudacellIds);
	thrust::device_ptr<int> thrustPIds = thrust::device_pointer_cast(cudapIds);
	thrust::sort_by_key(thrustCellIds, thrustCellIds+numParticles, thrustPIds);

	resetGrid<<<fullBlocksPerGridGrid, threadsPerBlockGrid>>>(int(gridSize.x*gridSize.y*gridSize.z), cudagrid);
	checkCUDAError("reset grid");
	setGridValuesKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(numParticles, int(gridSize.x*gridSize.y*gridSize.z), cudagrid, cudacellIds);
	checkCUDAError("set values in grid neighbor");
	
	cout<<"ERGWERTH"<<endl;

	findNeighborsUsingGridKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(numParticles, h, maxNeighbors, cudapositions, cudaneighbors, cudanumNeighbors, cudagrid, cudapIds, int(gridSize.x*gridSize.y*gridSize.z), cudaids, gridSize, cudacellIds);

	int* numNeighbors = new int[numParticles];
	hipMemcpy( numNeighbors, cudanumNeighbors, numParticles*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError(" copying num neighbor dbhtrwhnwe ");

	int avg = 0;
	for (int i=0; i<numParticles; i++){
		avg+=numNeighbors[i];
	}

	cout<<"average number of neighbors with grid: "<<float(avg)/float(numParticles)<<endl;

	delete [] numNeighbors;

	checkCUDAError(" finding neighbors using grid ");
}

void freeCudaGrid(){
	hipFree(cudapositions);
	hipFree(cudacellIds);
	hipFree(cudapIds);
	hipFree(cudagrid);
	hipFree(cudanumNeighbors);
	hipFree(cudaneighbors);
	hipFree(cudaids);
}

void test_uniform_grid(){

	int * a = new int[10];
	int * b = new int[10];

	for (int i=0; i<10; i+=1){
		a[i] = i;
		b[i] = i;
	}
	
	dim3 threadsPerBlock(16);
	dim3 fullBlocksPerGrid(64);

	int *cudaA, *cudaB;

	hipMalloc((void**)&cudaA, 10*sizeof(int));
	hipMalloc((void**)&cudaB, 10*sizeof(int));

	hipMemcpy(cudaA, a, 10*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, 10*sizeof(int), hipMemcpyHostToDevice);

	hello<<<fullBlocksPerGrid, threadsPerBlock>>>(cudaA, cudaB);

	hipMemcpy(a, cudaA, 10*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b, cudaB, 10*sizeof(int), hipMemcpyDeviceToHost);

	for (int i=0; i<10; i+=1){
		cout<<a[i]<<endl;
	}

	hipFree(cudaA);
	hipFree(cudaB);

	delete [] a;
	delete [] b;


}


hash_grid::hash_grid(int numParticles, glm::vec3* points, glm::vec3 gridSize){

	m_numParticles = numParticles;
	m_points = new glm::vec3[m_numParticles];
	m_ids = new int[m_numParticles];
	for (int i=0; i<m_numParticles; i+=1){
		m_points[i] = points[i];
		m_ids[i] = i;
	}

	m_gridSize = gridSize;

	c_positions = NULL;
	hipMalloc((void**)&c_positions, m_numParticles*sizeof(glm::vec3));
	hipMemcpy( c_positions, m_points, m_numParticles*sizeof(glm::vec3), hipMemcpyHostToDevice);
	
	c_ids = NULL;
	hipMalloc((void**)&c_ids, m_numParticles*sizeof(int));
	hipMemcpy( c_ids, m_ids, m_numParticles*sizeof(int), hipMemcpyHostToDevice);

	c_cellIds = NULL;
	hipMalloc((void**)&c_cellIds, m_numParticles*sizeof(int));

	c_pIds = NULL;
	hipMalloc((void**)&c_pIds, m_numParticles*sizeof(int));

	c_grid = NULL;
	hipMalloc((void**)&c_grid, int(m_gridSize.x*m_gridSize.y*m_gridSize.z)*sizeof(pair<int,int>));

	neighborsAlloc = false;
	m_maxNeighbors = -1;
}

void hash_grid::findNeighbors(int maxNeighbors, float h){
	if (maxNeighbors < 0){
		return;
	}

	if (m_maxNeighbors != maxNeighbors && m_maxNeighbors != -1){
		hipFree(c_numNeighbors);
		hipFree(c_neighbors);
		delete [] m_gridNeighbors;
		delete [] m_bruteNeighbors;
		delete [] m_gridNumNeighbors;
		delete [] m_bruteNumNeighbors;
	}

	m_maxNeighbors = maxNeighbors;
	m_h = h;

	c_neighbors = NULL;
	hipMalloc((void**)&c_neighbors, m_numParticles*m_maxNeighbors*sizeof(int*));

	c_numNeighbors = NULL;
	hipMalloc((void**)&c_numNeighbors, m_numParticles*sizeof(int));

	m_gridNeighbors = new int[m_numParticles*m_maxNeighbors];
	m_bruteNeighbors = new int[m_numParticles*m_maxNeighbors];
	m_gridNumNeighbors = new int[m_numParticles];
	m_bruteNumNeighbors = new int[m_numParticles];

	neighborsAlloc = true;

	dim3 threadsPerBlock(64);
	dim3 fullBlocksPerGrid(m_numParticles/8+1);

	dim3 threadsPerBlockGrid(64);
	dim3 fullBlocksPerGridGrid(int(m_gridSize.x*m_gridSize.y*m_gridSize.z)/8+1);


	hashParticlesToGridKernel<<<fullBlocksPerGrid, threadsPerBlock>>>
		(m_numParticles, c_cellIds, c_pIds, c_positions, m_gridSize, c_ids, h);
	hipDeviceSynchronize();
	
	checkCUDAError("hashing particles");

	thrust::device_ptr<int> thrustCellIds = thrust::device_pointer_cast(c_cellIds);
	thrust::device_ptr<int> thrustPIds = thrust::device_pointer_cast(c_pIds);
	thrust::sort_by_key(thrustCellIds, thrustCellIds+m_numParticles, thrustPIds);

	resetGrid<<<fullBlocksPerGridGrid, threadsPerBlockGrid>>>
		(int(m_gridSize.x*m_gridSize.y*m_gridSize.z), c_grid);
	
	checkCUDAError("reset grid");
	
	setGridValuesKernel<<<fullBlocksPerGrid, threadsPerBlock>>>
		(m_numParticles, int(m_gridSize.x*m_gridSize.y*m_gridSize.z), c_grid, c_cellIds);
	
	checkCUDAError("set values in grid neighbor");
	
	/////////////////////////
	resetNumNeighbors<<<fullBlocksPerGrid, threadsPerBlock>>>
		(m_numParticles, c_numNeighbors);
	findNeighborsUsingGridKernel<<<fullBlocksPerGrid, threadsPerBlock>>>
		(m_numParticles, h, m_maxNeighbors, c_positions, c_neighbors, c_numNeighbors, c_grid, 
		c_pIds, int(m_gridSize.x*m_gridSize.y*m_gridSize.z), c_ids, m_gridSize, c_cellIds);

	hipMemcpy( m_gridNumNeighbors, c_numNeighbors, m_numParticles*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError(" copying num neighbor dbhtrwhnwe ");

	hipMemcpy( m_gridNeighbors, c_neighbors, m_numParticles*m_maxNeighbors*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError(" copying neighbor dbhtrwhnwe ");

	int avg = 0;
	for (int i=0; i<m_numParticles; i++){
		avg+=m_gridNumNeighbors[i];
	}

	cout<<"average number of neighbors with grid: "<<float(avg)/float(m_numParticles)<<endl;

	//////////////////
	resetNumNeighbors<<<fullBlocksPerGrid, threadsPerBlock>>>
		(m_numParticles, c_numNeighbors);
	findNeighborsKernel<<<fullBlocksPerGrid, threadsPerBlock>>>
		(m_numParticles, c_positions, c_neighbors, c_numNeighbors, h, m_maxNeighbors, c_ids);

	hipMemcpy( m_bruteNumNeighbors, c_numNeighbors, m_numParticles*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError(" copying num neighbor dbhtrwhnwe ");

	hipMemcpy( m_bruteNeighbors, c_neighbors, m_numParticles*m_maxNeighbors*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError(" copying neighbor dbhtrwhnwe ");

	avg = 0;
	for (int i=0; i<m_numParticles; i++){
		avg+=m_bruteNumNeighbors[i];
	}

	cout<<"average number of neighbors with brute: "<<float(avg)/float(m_numParticles)<<endl;

	checkCUDAError(" finding neighbors using grid ");
}

int hash_grid::hashParticle(int id) const{
	glm::vec3 p = m_points[id];
	int x = p.x/m_h;
	int y = p.y/m_h;
	int z = p.z/m_h;
	return x + y*m_gridSize.x + z*m_gridSize.x*m_gridSize.y;
}

hash_grid::~hash_grid(){

	delete [] m_points;

	if (neighborsAlloc){
		hipFree(c_numNeighbors);
		hipFree(c_neighbors);
		delete [] m_gridNeighbors;
		delete [] m_bruteNeighbors;
		delete [] m_gridNumNeighbors;
		delete [] m_bruteNumNeighbors;
	}

	hipFree(c_positions);
	hipFree(c_cellIds);
	hipFree(c_pIds);
	hipFree(c_grid);
	hipFree(c_ids);
}