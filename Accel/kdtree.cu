#include "kdtree.h"


kdtree::kdtree(mesh* m){
	this->m_mesh = m;
	
	this->cudaTris = NULL;
	hipMalloc((void**)&cudaTris, m->numTris*sizeof(glm::vec3));

	this->cudaVerts = NULL;
	hipMalloc((void**)&cudaVerts, m->numVerts*sizeof(glm::vec3));


	//debug
	this->boundingBoxes = new boundingBox[m->numTris];
	cudaBoundingBoxes = NULL;
	hipMalloc((void**)&cudaBoundingBoxes, m->numTris*sizeof(boundingBox));
}

kdtree::~kdtree(){
	
	hipFree(this->cudaTris);
	hipFree(this->cudaVerts);

	delete [] this->boundingBoxes;
	hipFree(this->cudaBoundingBoxes);
}